
#include "hip/hip_runtime.h"

#include <stdio.h>

#define N (1024 * 64)

__global__ void add(int* a, int* b, int* c) 
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) 
  {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}


int main()
{
  int a[N];
  int b[N];
  int c[N];
  int* dev_a;
  int* dev_b;
  int* dev_c;
  // Allocate memory for the GPU arrays
  hipMalloc(&dev_a, N * sizeof(int));
  hipMalloc(&dev_b, N * sizeof(int));
  hipMalloc(&dev_c, N * sizeof(int));

  // Fill a and b with some "random" numbers
  for (int i = 0; i < N; i++)
  {
    a[i] = i;
    b[i] = i * i;
  }
  // Copy a and b to the gpu
  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  // Perform the addition
  add<<<128, 128>>>(dev_a, dev_b, dev_c);

  // Copy back the result to host
  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

  // Verify calculation
  bool success = true;
  for (int i = 0; i < N; i++)
  {
    if ((a[i] + b[i]) != c[i])
    {
      printf("Error at index: %d, %d + %d != %d\n", i, a[i], b[i], c[i]);
      success = false;
    }
  }

  if (success)
  {
    printf("Vector addition successful!\n");
  }

  //free memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}